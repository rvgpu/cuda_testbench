#include "hip/hip_runtime.h"
#include <cstdio>
#include <gtest/gtest.h>

__global__ void add(float *x, float *y, float *out)
{
    out[threadIdx.x] = x[threadIdx.x] + y[threadIdx.x];
}

__global__ void sub(float *x, float *y, float *out)
{
    out[threadIdx.x] = x[threadIdx.x] - y[threadIdx.x];
}

__global__ void mul(float *x, float *y, float *out)
{
    out[threadIdx.x] = x[threadIdx.x] * y[threadIdx.x];
}

__global__ void div(float *x, float *y, float *out)
{
    out[threadIdx.x] = x[threadIdx.x] / y[threadIdx.x];
}

class CUDATestOfArrayFloat : public ::testing::Test {
protected:
    void SetUp() override {
        kDataLen    = 64;
        in1         = new float[kDataLen];
        in2         = new float[kDataLen];
        result      = new float[kDataLen];
        reference   = new float[kDataLen];

        for (int i=0; i<kDataLen; ++i) {
            in1[i] = float(i);
            in2[i] = float(i) + 100.0f;
            result[i] = -1.0f;
            reference[i] = -2.0f;
        }

        hipMalloc(&device_in1, kDataLen * sizeof(float));
        hipMalloc(&device_in2, kDataLen * sizeof(float));
        hipMalloc(&device_out, kDataLen * sizeof(float));

        hipMemcpy(device_in1, in1, kDataLen * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_in2, in2, kDataLen * sizeof(float), hipMemcpyHostToDevice);
    }

    void TearDown() override {
        delete in1;
        delete in2;
        delete result;
        delete reference;

        hipDeviceReset();
    }

    int kDataLen;
    float *in1;
    float *in2;
    float *result;
    float *reference;

    float *device_in1;
    float *device_in2;
    float *device_out;
};

// int main()
TEST_F(CUDATestOfArrayFloat, Add)
{
    add<<<1, kDataLen>>>(device_in1, device_in2, device_out);
    // Copy output data to host
    hipDeviceSynchronize();
    hipMemcpy(result, device_out, kDataLen * sizeof(float), hipMemcpyDeviceToHost);

    // print the results
    for (int i=0; i<kDataLen; ++i) {
        reference[i] = in1[i] + in2[i];
        EXPECT_FLOAT_EQ(result[i], reference[i]);
    }
}

TEST_F(CUDATestOfArrayFloat, Sub)
{
    sub<<<1, kDataLen>>>(device_in1, device_in2, device_out);
    // Copy output data to host
    hipDeviceSynchronize();
    hipMemcpy(result, device_out, kDataLen * sizeof(float), hipMemcpyDeviceToHost);

    // print the results
    for (int i=0; i<kDataLen; ++i) {
        reference[i] = in1[i] - in2[i];
        EXPECT_FLOAT_EQ(result[i], reference[i]);
    }
}

TEST_F(CUDATestOfArrayFloat, Mul)
{
    mul<<<1, kDataLen>>>(device_in1, device_in2, device_out);
    // Copy output data to host
    hipDeviceSynchronize();
    hipMemcpy(result, device_out, kDataLen * sizeof(float), hipMemcpyDeviceToHost);

    // print the results
    for (int i=0; i<kDataLen; ++i) {
        reference[i] = in1[i] * in2[i];
        EXPECT_FLOAT_EQ(result[i], reference[i]);
    }
}

TEST_F(CUDATestOfArrayFloat, Div)
{
    div<<<1, kDataLen>>>(device_in1, device_in2, device_out);
    // Copy output data to host
    hipDeviceSynchronize();
    hipMemcpy(result, device_out, kDataLen * sizeof(float), hipMemcpyDeviceToHost);

    // print the results
    for (int i=0; i<kDataLen; ++i) {
        reference[i] = in1[i] / in2[i];
        EXPECT_FLOAT_EQ(result[i], reference[i]);
    }
}